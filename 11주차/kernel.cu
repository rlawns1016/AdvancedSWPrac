#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

//1 exercise //1-2
//2 prac 1-3	//1-3
//3 Fibonacci	//HW
//4 gen	//1-3 ����� �ʿ��� ���� ����
#define prac 2


#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;
#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1
#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

#if prac==1



typedef struct {
	int width;
	int height;
	float *elements;
} Array;


#define MAX_N_ELEMENTS	(1 << 26) // �������� ���� (1 << 20) = 2^20

//GPU : GeForce GT 635
//Compute Capability : 2.1
//SPEC : https://www.geforce.com/hardware/desktop-gpus/geforce-gt-635-oem/specifications
//GPU Memory : 2048MB
//2048 / 3 (��� 3��)

void generate_random_float_array(float *array, int n) {

	int i;

	for (i = 0; i < n; i++) {
		array[i] = 3.1415926f*((float)rand() / RAND_MAX);
	}
}
void combine_two_arrays(float *x, float *y, float *z, int n) {
	int i;
	//CPU ����
	for (i = 0; i < n; i++) {
		z[i] = 1.0f / (sin(x[i])*cos(y[i]) + cos(x[i])*sin(y[i]));
	}
}

__global__ void CombineTwoArrraysKernel(Array A, Array B, Array C) {
	int row = blockDim.y*blockIdx.y + threadIdx.y;
	int col = blockDim.x*blockIdx.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;
	C.elements[id] = 1.0f / (sin(A.elements[id])*cos(B.elements[id])+ cos(A.elements[id])*sin(B.elements[id]));
}

hipError_t combine_two_arrays_GPU(const Array A, const Array B, Array C);

int BLOCK_SIZE = 16;

int main()
{
	int n_elements;

	srand((unsigned int)time(NULL));
	n_elements = MAX_N_ELEMENTS;
	Array A, B, C, G;
	A.width = B.width = C.width = G.width =1024;
	A.height = B.height = C.height = G.height = MAX_N_ELEMENTS / 1024;

	A.elements = (float *)malloc(sizeof(float)*MAX_N_ELEMENTS);
	B.elements = (float *)malloc(sizeof(float)*MAX_N_ELEMENTS);
	C.elements = (float *)malloc(sizeof(float)*MAX_N_ELEMENTS);
	G.elements = (float *)malloc(sizeof(float)*MAX_N_ELEMENTS);
	generate_random_float_array(A.elements, MAX_N_ELEMENTS);
	generate_random_float_array(B.elements, MAX_N_ELEMENTS);
	
	CHECK_TIME_START;
	combine_two_arrays(A.elements, B.elements, C.elements, n_elements);
	CHECK_TIME_END(compute_time);

	printf("***CPU C[10] = %f/ Time taken = %.6fms\n", C.elements[10], compute_time);


	hipError_t cudaStatus = combine_two_arrays_GPU(A, B, G);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "combine_two_arrays_GPU failed!");
        return 1;
    }

	printf("***GPU G[10] = %f/ Time taken = %.6fms\n", G.elements[10], device_time);
	//7ms����

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
hipError_t combine_two_arrays_GPU(const Array A, const Array B, Array C) {
	
	//�Ʒ� �Լ����� ����Ͽ� ��� �ϸ� ������ ��Ȯ�� �ð��� ������ �� ������ �����غ� ��.
	//���� ���κи� �ð� ����
	CHECK_TIME_INIT_GPU()
	//CHECK_TIME_START_GPU()
	//CHECK_TIME_END_GPU(device_time)
	//CHECK_TIME_DEST_GPU()
	
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}/////////////  if(cu.....  ==CUDA_CALL
	
	Array d_A, d_B, d_C;
	size_t size;

	d_A.width = A.width; d_A.height = A.height;
	size = A.width * A.height * sizeof(float);
	CUDA_CALL(hipMalloc(&d_A.elements, size))	//GPU�� �޸𸮿� �Ҵ�
	
	CUDA_CALL(hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice)) //main memory -> GPU memory

	d_B.width = B.width; d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	CUDA_CALL(hipMalloc(&d_B.elements, size))
	CUDA_CALL(hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice))

	d_C.width = C.width; d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	CUDA_CALL(hipMalloc(&d_C.elements, size))

	// Assume that width and height are multiples of BLOCK SIZE.
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);	//���� ���� ����ص� ��
	dim3 dimGrid(A.width / dimBlock.x, A.height / dimBlock.y);

	CHECK_TIME_START_GPU()
	CombineTwoArrraysKernel <<< dimGrid, dimBlock >>> (d_A, d_B, d_C);	//cuda complier�� ó��
	CHECK_TIME_END_GPU(device_time)

	CUDA_CALL(hipGetLastError())

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	CUDA_CALL(hipDeviceSynchronize())
	CUDA_CALL(hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost))	//GPU memory -> main memory
	

Error:
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
	CHECK_TIME_DEST_GPU()
	return cudaStatus;
}
#endif

#if prac==2

int n;
#define BLOCK_SIZE 32
const int ELEM_PER_VECTOR = 32;
float (*pVecX)[ELEM_PER_VECTOR], (*pVecY)[ELEM_PER_VECTOR], (*pVecY_G)[ELEM_PER_VECTOR];
float(*pMatA)[ELEM_PER_VECTOR];
void init_MatVec(void)
{
	int i, j;
	srand((unsigned)time(NULL));
	FILE* fp = fopen("gen.bin", "rb");
	fread(&n, sizeof(float), 1, fp);

	pVecX = new float[n][ELEM_PER_VECTOR];
	pVecY = new float[n][ELEM_PER_VECTOR];
	pVecY_G = new float[n][ELEM_PER_VECTOR];
	pMatA = new float[ELEM_PER_VECTOR][ELEM_PER_VECTOR];
	

	fread(pVecX, sizeof(float), n * ELEM_PER_VECTOR, fp);
	fread(pMatA, sizeof(float), ELEM_PER_VECTOR * ELEM_PER_VECTOR, fp);

	for(i = 0; i < n; i++)
	{
		for(j = 0; j < ELEM_PER_VECTOR; j++)
		{
			pVecY[i][j] = 0;
			pVecY_G[i][j] = 0;
		}
	}

	fclose(fp);
}
void Mat_Vec_Multiply()
{
	//CPU
	int i, j, k;

	for(i = 0; i < n; i++)
	{
		for(j = 0; j < ELEM_PER_VECTOR; j++)
		{
			for(k = 0; k < ELEM_PER_VECTOR; k++)
			{
				pVecY[i][j] += pMatA[j][k] * pVecX[i][k];
			}
		}
	}
}

__global__ void Mat_Vec_Multiply_Kernel(float * A, float * B, float *C)
{ 
	//GPU
	int row = blockDim.y*blockIdx.y + threadIdx.y;
	int col = blockDim.x*blockIdx.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;

	C[id] = 0;

	for(int i = 0; i < ELEM_PER_VECTOR; i++)
		C[id] += A[col * ELEM_PER_VECTOR + i] * B[row * ELEM_PER_VECTOR + i];
}

hipError_t combine_two_arrays_GPU(const float (*pMatA)[ELEM_PER_VECTOR], const float (*pVecX)[ELEM_PER_VECTOR], float (*pVecY_G)[ELEM_PER_VECTOR])
{
	float *A, *G, *X;
	size_t size = ELEM_PER_VECTOR * ELEM_PER_VECTOR * sizeof(float);
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	//CUDA_CALL(hipSetDevice(0))
	
	CUDA_CALL(hipMalloc(&A, size))
	size = ELEM_PER_VECTOR * n * sizeof(float);
	CUDA_CALL(hipMalloc(&X, size))
	CUDA_CALL(hipMalloc(&G, size))

	size = ELEM_PER_VECTOR * ELEM_PER_VECTOR * sizeof(float);
	CUDA_CALL(hipMemcpy(A, pMatA, size, hipMemcpyHostToDevice))
	size = ELEM_PER_VECTOR * n * sizeof(float);
	CUDA_CALL(hipMemcpy(X, pVecX, size, hipMemcpyHostToDevice))
	

	dim3 dimBlock(1, BLOCK_SIZE);	//���� ���� ����ص� ��
	dim3 dimGrid(BLOCK_SIZE, n / BLOCK_SIZE);

	CHECK_TIME_INIT_GPU()
	CHECK_TIME_START_GPU();
	Mat_Vec_Multiply_Kernel <<< dimGrid, dimBlock >>> (A, X, G);
	CHECK_TIME_END_GPU(device_time);

	CUDA_CALL(hipGetLastError())
	

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	CUDA_CALL(hipDeviceSynchronize())
	CUDA_CALL(hipMemcpy(pVecY_G, G, size, hipMemcpyDeviceToHost))

Error:
	hipFree(A);
	hipFree(X);
	hipFree(G);
	CHECK_TIME_DEST_GPU()
	return cudaStatus;
}

int main()
{
	init_MatVec();
	printf("n = %d  file open ok.\n", n);

	
	CHECK_TIME_START;
	Mat_Vec_Multiply();
	CHECK_TIME_END(compute_time);
	printf("***CPU C[10] = %.3f/ Time taken = %.6fms\n", pVecY[0][0], compute_time);

	hipError_t cudaStatus = combine_two_arrays_GPU(pMatA, pVecX, pVecY_G);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "combine_two_arrays_GPU failed!");
        return 1;
  	}
	
	printf("***GPU C[10] = %.3f/ Time taken = %.6fms\n", pVecY_G[0][0], device_time);
	
	delete pVecX;
	delete pVecY;
	delete pVecY_G;
	delete pMatA;

	cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}


#endif

#if prac==3

#define N 67108864 // 8192 * 8192 = 2^13 * 2^13
int Fibonacci(int n) {
	// DO NOT MODIFY THIS FUNCTION!!!
	float sqrt_5, x_0, x_1;
	float tmp_0, tmp_1;

	sqrt_5 = sqrtf(5.0f);
	x_0 = (1.0f + sqrt_5) / 2.0f;
	x_1 = (1.0f - sqrt_5) / 2.0f;

	tmp_0 = tmp_1 = 1.0f;
	for (int i = 0; i < n; i++) {
		tmp_0 *= x_0;
		tmp_1 *= x_1;
	}
	return (int)((tmp_0 - tmp_1) / sqrt_5 + 0.5);
}
void generate_input(int *x, int n) {
	// DO NOT MODIFY THIS FUNCTION!!!
	srand((unsigned int)time(NULL));

	for (int i = 0; i < n; i++) {
		x[i] = 35 + (int)(5.0f * rand() / RAND_MAX + 0.5f);
	}
}
__global__ void Fibonacci_Kernel(int *x, int *y)
{
	int row = blockDim.y*blockIdx.y + threadIdx.y;
	int col = blockDim.x*blockIdx.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;
	//int id = threadIdx.x;
	float sqrt_5, x_0, x_1;
	float tmp_0, tmp_1;

	sqrt_5 = sqrtf(5.0f);
	x_0 = (1.0f + sqrt_5) / 2.0f;
	x_1 = (1.0f - sqrt_5) / 2.0f;

	tmp_0 = tmp_1 = 1.0f;
	for (int i = 0; i < x[id]; i++) {
		tmp_0 *= x_0;
		tmp_1 *= x_1;
	}
	y[id] = (int)((tmp_0 - tmp_1) / sqrt_5 + 0.5);
}
void Fibonacci_GPU(int *x, int *y)
{
	hipError_t cudaStatus;
		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

}

int *x, *y_c, *y_g; // input/output arrays

void main(void) {
	
	int n, i;
	FILE *fp;

	// Read the input array from the input file if one already exists.
	fp = fopen("x.binary", "rb");
	if (!fp) {
		fprintf(stderr, "Error: cannot open the input file...\n");
		exit(-1);
	}
	fread(&n, sizeof(int), 1, fp);
	fprintf(stdout, "\n*** The problem size is %d.\n", n);

	x = (int *)malloc(sizeof(int)*n);
	if (!x) {
		fprintf(stderr, "Error: cannot allocate memory for the input array...\n");
		exit(-1);
	}
	fread(x, sizeof(int), n, fp);
	fclose(fp);

	y_c = (int *)malloc(sizeof(int)*n); 
	y_g = (int *)malloc(sizeof(int)*n);

	//CPU
	CHECK_TIME_START;
	for (i = 0; i < n; i++) {
		y_c[i] = Fibonacci(x[i]);
	}
	CHECK_TIME_END(compute_time);
	//GPU
	Fibonacci_GPU(x, y_g);


	fprintf(stdout, "\n***_CPU_ Time taken for computing %d Fibonacci numbers is %.6fms\n\n", n, compute_time);

	//fprintf(stdout, "\n***_GPU_ Time taken for computing %d Fibonacci numbers is %.6fms\n\n", n, device_time);

	i = (int)(n * (rand() / (RAND_MAX + 1.0f)));
	fprintf(stdout, "*** Fibonacci number of %d is (CPU :%d  , GPU :%d).\n\n", x[i], y_c[i], y_g[i]);

	// Write the output array into the output file.
	fp = fopen("y.binary", "wb");
	if (!fp) {
		fprintf(stderr, "Error: cannot open the output file...\n");
		exit(-1);
	}
	fwrite(&n, sizeof(int), 1, fp);
	fwrite(y_c, sizeof(int), n, fp);
	fclose(fp);

	free(x);
	free(y_c);
	free(y_g);
}


#endif

#if prac==4

const int ELEM_PER_VECTOR = 32;

int main()
{
	int n;

	srand((unsigned)&n);

	printf("Enter a size: ");
	scanf("%d", &n);

	int size = ELEM_PER_VECTOR * n;
	float* vec = new float[size];

	for (int i = 0; i < size; ++i)
	{
		vec[i] = (float(rand()) * 2.f / RAND_MAX) - 1.f;
	}

	float(*mat)[ELEM_PER_VECTOR] = new float[ELEM_PER_VECTOR][ELEM_PER_VECTOR];
	for (int i = 0; i < ELEM_PER_VECTOR; ++i)
	{
		for (int j = 0; j < ELEM_PER_VECTOR; ++j)
		{
			mat[i][j] = (float(rand()) * 2.f / RAND_MAX) - 1.f;
		}
	}

	FILE* fp = fopen("gen.bin", "wb");
	fwrite(&n, sizeof(float), 1, fp);
	fwrite(vec, sizeof(float), size, fp);
	fwrite(mat, sizeof(float), ELEM_PER_VECTOR * ELEM_PER_VECTOR, fp);
	fclose(fp);

	fp = fopen("gen.bin", "rb");
	float* vec2 = new float[size];
	float(*mat2)[ELEM_PER_VECTOR] = new float[ELEM_PER_VECTOR][ELEM_PER_VECTOR];
	int m;
	fread(&m, sizeof(float), 1, fp);
	fread(vec2, sizeof(float), m * ELEM_PER_VECTOR, fp);
	fread(mat2, sizeof(float), ELEM_PER_VECTOR * ELEM_PER_VECTOR, fp);

	if (n != m) printf("error: size diff. %n != %n", n, m);
	for (int i = 0; i < size; ++i)
	{
		if (vec[i] != vec2[i])
		{
			printf("[%d] %f != %f\n", vec[i], vec2[i]);
			break;
		}
	}
	for (int i = 0; i < ELEM_PER_VECTOR; ++i)
	{
		for (int j = 0; j < ELEM_PER_VECTOR; ++j)
		{
			if (mat[i][j] != mat2[i][j])
			{
				printf("[%d][%d]\n", i, j);
				break;
			}
		}
	}
	fclose(fp);

	delete[] vec;

	return 0;
}

#endif